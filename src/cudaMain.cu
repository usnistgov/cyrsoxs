#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////
// MIT License
//
//Copyright (c) 2019 - 2020 Iowa State University
//
//Permission is hereby granted, free of charge, to any person obtaining a copy
//of this software and associated documentation files (the "Software"), to deal
//in the Software without restriction, including without limitation the rights
//to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
//copies of the Software, and to permit persons to whom the Software is
//furnished to do so, subject to the following conditions:
//
//The above copyright notice and this permission notice shall be included in all
//copies or substantial portions of the Software.
//
//THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
//IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
//FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
//AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
//LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
//OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
//SOFTWARE.
//////////////////////////////////////////////////////////////////////////////////


#include <cudaMain.h>
#include <Input/Input.h>
#include <bits/ios_base.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <Output/writeVTI.h>
#include <uniaxial.h>
#include <hipblas.h>
#include <chrono>
#include <ctime>
#include <chrono>
#include <npp.h>
#include <Output/outputUtils.h>
#include <stdint.h>  // for UINT32_MAX

#define START_TIMER(X) timerArrayStart[X] = std::chrono::high_resolution_clock::now();
#define END_TIMER(X) timerArrayEnd[X] = std::chrono::high_resolution_clock::now(); \
                     timings[X] +=  (static_cast<std::chrono::duration<Real>>(timerArrayEnd[X] - timerArrayStart[X])).count();


int warmup(){
  double *d_warmup, *warmup;
  warmup = new double[1000];
  CUDA_CHECK_RETURN(hipMalloc((void **) &d_warmup, sizeof(double) * 1000));
  gpuErrchk(hipPeekAtLastError());
  CUDA_CHECK_RETURN(hipMemcpy(d_warmup, warmup, sizeof(double) * 1000, hipMemcpyHostToDevice));
  gpuErrchk(hipPeekAtLastError());
  hipFree(d_warmup);
  delete[] warmup;
  return EXIT_SUCCESS;
}





__global__ void computePolarization(Material<NUM_MATERIAL> materialInput,
                                    const Voxel<NUM_MATERIAL> *voxelInput,
                                    const ElectricField elefield,
                                    const Real angle,
                                    const uint3 voxel,
                                    Complex *polarizationX,
                                    Complex *polarizationY,
                                    Complex *polarizationZ,
                                    FFT::FFTWindowing windowing,
                                    const bool enable2D
) {
  UINT threadID = threadIdx.x + blockIdx.x * blockDim.x;
  const UINT voxelNum = voxel.x*voxel.y*voxel.z;

  if (threadID >= voxelNum) {
    return;
  }

#ifndef BIAXIAL
    computePolarizationUniaxial(&materialInput,angle,voxelInput,threadID,polarizationX,polarizationY,polarizationZ);
#else
    printf("Kernel not spported\n");
#endif


if(windowing == FFT::FFTWindowing::HANNING) {
  UINT Z = static_cast<UINT>(floorf(threadID / (voxel.y * voxel.x * 1.0)));
  UINT Y = static_cast<UINT>(floorf((threadID - Z * voxel.y * voxel.x) / (voxel.x * 1.0)));
  UINT X = static_cast<UINT>(threadID - Y * voxel.x - Z * voxel.y * voxel.x);
  Real3 hanningWeight;
  hanningWeight.x = static_cast<Real> (0.5 * (1 - cos(2 * M_PI * X / (voxel.x))));
  hanningWeight.y = static_cast<Real> (0.5 * (1 - cos(2 * M_PI * Y / (voxel.y))));
  hanningWeight.z = static_cast<Real>(1.0);
  if(not(enable2D)){
    hanningWeight.z = static_cast<Real>(0.5 * (1 - cos(2 * M_PI * Z / (voxel.z))));
  }
  Real totalHanningWeight = hanningWeight.x * hanningWeight.y * hanningWeight.z;
  polarizationX[threadID].x *= totalHanningWeight;
  polarizationX[threadID].y *= totalHanningWeight;
  polarizationY[threadID].x *= totalHanningWeight;
  polarizationY[threadID].y *= totalHanningWeight;
  polarizationZ[threadID].x *= totalHanningWeight;
  polarizationZ[threadID].y *= totalHanningWeight;

}

}


/**
 * Return false if system is too large to handle
 *
 * This checks whether the system is large enough to require
 * 64 bit numbers for the indices. If so, it also checks whether
 * data types will support this.
 *
 * @param voxel dimensions of voxel data
 *
 * @return false if system size cannot be supported
 */
bool check_system_size(const UINT *voxel) {
  if (static_cast<uint64_t>(voxel[0]) * voxel[1] * voxel[2] > UINT32_MAX) {
    // need 64 bit sizes
    if (sizeof(BigUINT) < 8) {
      // BigUNIT needs to be set to 64-bit in include/DataTypes.h
      // in order for this to work.
      return false;
    } else {
      // There are currently multiple places in the code where this is
      // implemented incorrectly. Even if fixed, it will run into memory
      // limits.
      std::cout << "[WARNING] large system size implementation may be wrong\n";
    }
  }
  return true;
}


int cudaMain(const UINT *voxel,
             const InputData &idata,
             const std::vector<Material<NUM_MATERIAL> > &materialInput,
             Real * projectionGPUAveraged,
             const Voxel<NUM_MATERIAL> *voxelInput) {
  
  // check if system size is greater than data types can handle
  if (!check_system_size(voxel)) {
    std::cerr << "[ERROR] System is too large for compiled options\n";
    std::cerr << "[ERROR] This must be fixed in the code\n";
    return (EXIT_FAILURE);
  }

  const BigUINT voxelSize = voxel[0] * voxel[1] * voxel[2]; /// Voxel size
  const UINT
      numAnglesRotation = static_cast<UINT>(std::round((idata.endAngle - idata.startAngle) / idata.incrementAngle + 1));
  const UINT numEnergyLevel = static_cast<UINT>(idata.energies.size());


  int num_gpu;
  hipGetDeviceCount(&num_gpu);
  std::cout << "Number of CUDA devices:" << num_gpu << "\n";

  if(num_gpu < 1){
    std::cout << "No GPU found. Exiting" << "\n";
    return (EXIT_FAILURE);
  }




#ifdef PROFILING
  enum TIMERS:UINT{
    MALLOC = 0,
    MEMCOPY_CPU_GPU = 1,
    POLARIZATION = 2,
    FFT = 3,
    SCATTER3D = 4,
    EWALDS = 5,
    ROTATION=6,
    MEMCOPY_GPU_CPU = 7,
    ENERGY=8,
    MAX = 9
  };
  static const char *timersName[]{"Malloc on CPU + GPU",
                                  "Memcopy CPU -> GPU",
                                  "Polarization",
                                  "FFT",
                                  "Scatter3D",
                                  "Ewalds",
                                  "Rotation",
                                  "Memcopy GPU -> CPU",
                                  "Total time "};
  static_assert(sizeof(timersName) / sizeof(char*) == TIMERS::MAX,
                "sizes dont match");
  std::array<std::chrono::high_resolution_clock::time_point,TIMERS::MAX> timerArrayStart;
  std::array<std::chrono::high_resolution_clock::time_point,TIMERS::MAX> timerArrayEnd;
  std::array<Real,TIMERS::MAX> timings{};
  timings.fill(0.0);

#endif

#ifdef DUMP_FILES
  createDirectory("Polarize");
  createDirectory("FFT");
  createDirectory("Scatter");

  /** Writing VTI files as a cross check **/

#if (NUM_MATERIAL==2)
  const char * varnameVector[2] = {"material1_s1","material2_s1"};
#elif (NUM_MATERIAL==4)
  const char * varnameVector[4] = {"material1_s","material2_s","material3_s","material4_s"};
#endif
#if (NUM_MATERIAL==2)
  const char * varnameScalar[2] = {"phi0","phi1"};
#elif (NUM_MATERIAL==4)
  const char * varnameScalar[4] = {"phi0","phi1", "phi2", "phi3"};
#endif

  VTI::writeVoxelDataVector(voxelInput, voxel, "S1", varnameVector);
  VTI::writeVoxelDataScalar(voxelInput, voxel, "Phi", varnameScalar);
#endif

  omp_set_num_threads(num_gpu);
#pragma omp parallel
  {


    int device_gpu = -1;
    hipSetDevice(omp_get_thread_num());
    hipDeviceProp_t dprop;
    hipGetDeviceProperties(&dprop, omp_get_thread_num());
    hipGetDeviceCount(&device_gpu);
#ifdef PROFILING
    if(warmup() == EXIT_SUCCESS){
      std::cout << "Warmup completed on GPU " << dprop.name << "\n";
    }
    else{
      std::cout << "Warmup failed on GPU " << dprop.name << "\n";
#pragma omp cancel parallel
      exit (EXIT_FAILURE);
    }
#endif

    hipfftResult result;
    hipfftHandle plan;
#ifdef DOUBLE_PRECISION
    hipfftPlan3d(&plan, voxel[2], voxel[1], voxel[0], HIPFFT_Z2Z);
#else
    hipfftPlan3d(&plan, voxel[2], voxel[1], voxel[0], HIPFFT_C2C);
#endif
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    hipblasCreate(&handle);

    NppiSize sizeImage;
    sizeImage.height = voxel[0];
    sizeImage.width = voxel[1];

    NppiRect rect;
    rect.height = voxel[0];
    rect.width = voxel[1];
    rect.x = 0;
    rect.y = 0;

    int omp_thread_id = omp_get_thread_num();
    // index of last energy for this thread (for infor output below)
    int last_energy_idx = (numEnergyLevel / num_gpu - 1) * num_gpu + omp_thread_id;
    if (last_energy_idx + num_gpu < numEnergyLevel) {
      last_energy_idx += num_gpu;
    }

    if(last_energy_idx > numEnergyLevel){
      std::cout << "[INFO] [GPU = " << dprop.name  << "] -> No computation. Idle\n";
    }
    else{
      std::cout << "[INFO] [GPU = " << dprop.name  << "] : " << idata.energies[omp_thread_id] << "eV -> " << idata.energies[last_energy_idx] << "eV\n" ;
    }


#ifdef PROFILING
    {
      START_TIMER(TIMERS::MALLOC);
    }
#endif
    uint3 vx;
    vx.x = voxel[0];
    vx.y = voxel[1];
    vx.z = voxel[2];
    Complex *polarizationZ = new Complex[voxelSize];
    Complex *polarizationX = new Complex[voxelSize];
    Complex *polarizationY = new Complex[voxelSize];
    Real *scatter3D = new Real[voxelSize];

#ifdef EOC
    Real *projectionCPU = new Real[BATCH * voxel[0] * voxel[1]];
#else

#endif

    Complex *d_polarizationZ, *d_polarizationX, *d_polarizationY;
    Real *d_scatter3D;
    UINT * d_mask;
    CUDA_CHECK_RETURN(hipMalloc((void **) &d_polarizationZ, sizeof(Complex) * voxelSize));
    gpuErrchk(hipPeekAtLastError());
    CUDA_CHECK_RETURN(hipMalloc((void **) &d_polarizationX, sizeof(Complex) * voxelSize));
    gpuErrchk(hipPeekAtLastError());
    CUDA_CHECK_RETURN(hipMalloc((void **) &d_polarizationY, sizeof(Complex) * voxelSize));
    gpuErrchk(hipPeekAtLastError());
    CUDA_CHECK_RETURN(hipMalloc((void **) &d_scatter3D, sizeof(Real) * voxelSize));
    gpuErrchk(hipPeekAtLastError());



#ifndef EOC
    Real *d_projection, *d_rotProjection, *d_projectionAverage;
    CUDA_CHECK_RETURN(hipMalloc((void **) &d_projection, sizeof(Real) * (voxel[0] * voxel[1])));
    gpuErrchk(hipPeekAtLastError());
    CUDA_CHECK_RETURN(hipMalloc((void **) &d_rotProjection, sizeof(Real) * (voxel[0] * voxel[1])));
    gpuErrchk(hipPeekAtLastError());
    if(idata.rotMask){
      CUDA_CHECK_RETURN(hipMalloc((void **) &d_mask, sizeof(UINT) * voxel[0]*voxel[1]));
      gpuErrchk(hipPeekAtLastError());
    }

    CUDA_CHECK_RETURN(hipMalloc((void **) &d_projectionAverage, sizeof(Real) * (voxel[0] * voxel[1])));
    gpuErrchk(hipPeekAtLastError());

#endif

    Voxel<NUM_MATERIAL> *d_voxelInput;
    CUDA_CHECK_RETURN(hipMalloc((void **) &d_voxelInput, sizeof(Voxel<NUM_MATERIAL>) * voxelSize));
    gpuErrchk(hipPeekAtLastError());

#ifdef PROFILING
    {
      END_TIMER(TIMERS::MALLOC)
      START_TIMER(TIMERS::MEMCOPY_CPU_GPU)
    }
#endif

    CUDA_CHECK_RETURN(hipMemcpy(d_voxelInput,
                                 voxelInput,
                                 sizeof(Voxel<NUM_MATERIAL>) * voxelSize,
                                 hipMemcpyHostToDevice));
    gpuErrchk(hipPeekAtLastError());

#ifdef PROFILING
    {
      END_TIMER(TIMERS::MEMCOPY_CPU_GPU)
    }
#endif

    UINT BlockSize = static_cast<UINT >(ceil(voxelSize * 1.0 / NUM_THREADS));
    UINT BlockSize2 = static_cast<UINT>(ceil(voxel[0] * voxel[1] * 1.0 / NUM_THREADS));

    for (UINT j = omp_thread_id; j < idata.energies.size(); j+=num_gpu) {

      Real energy = idata.energies[j];
      std::cout << " [STAT] Energy = " << energy << " starting "  << "\n";

      CUDA_CHECK_RETURN(hipMemset(d_projectionAverage, 0, voxel[0] * voxel[1] * sizeof(Real)));
      gpuErrchk(hipPeekAtLastError());
      if(idata.rotMask) {
        hipMemset(d_mask, 0, sizeof(UINT) * voxel[0] * voxel[1]);
      }

#ifdef  PROFILING
      START_TIMER(TIMERS::ENERGY)
#endif

      ElectricField eleField;
      eleField.e.x = 1;
      eleField.e.y = 0;
      eleField.e.z = 0;
      Real wavelength = static_cast<Real>(1239.84197 / energy);
      eleField.k.x = 0;
      eleField.k.y = 0;
      eleField.k.z = static_cast<Real>(2 * M_PI / wavelength);;
      Real angle;
      for (int i = 0; i < numAnglesRotation; i++) {
        angle = static_cast<Real>((idata.startAngle + i*idata.incrementAngle) * M_PI / 180.0);
#ifdef PROFILING
        {
          START_TIMER(TIMERS::POLARIZATION)
        }
#endif

        computePolarization <<< BlockSize, NUM_THREADS >>> (materialInput[j], d_voxelInput, eleField, angle, vx, d_polarizationX, d_polarizationY, d_polarizationZ,
                static_cast<FFT::FFTWindowing >(idata.windowingType), idata.if2DComputation());

        gpuErrchk(hipPeekAtLastError());
        hipDeviceSynchronize();
#ifdef DUMP_FILES

        CUDA_CHECK_RETURN(hipMemcpy(polarizationX,
                                     d_polarizationX,
                                     sizeof(Complex) * voxelSize,
                                     hipMemcpyDeviceToHost));
        gpuErrchk(hipPeekAtLastError());
        CUDA_CHECK_RETURN(hipMemcpy(polarizationZ,
                                     d_polarizationZ,
                                     sizeof(Complex) * voxelSize,
                                     hipMemcpyDeviceToHost));
        gpuErrchk(hipPeekAtLastError());
        CUDA_CHECK_RETURN(hipMemcpy(polarizationY,
                                     d_polarizationY,
                                     sizeof(Complex) * voxelSize,
                                     hipMemcpyDeviceToHost));
        gpuErrchk(hipPeekAtLastError());
        {
          std::string dirname = "Polarize/";
          std::string fname = dirname + "polarizationX" + std::to_string(i);
          VTI::writeDataScalar(polarizationX, voxel, fname.c_str(), "polarizeX");
          fname = dirname + "polarizationY" + std::to_string(i);
          VTI::writeDataScalar(polarizationY, voxel, fname.c_str(), "polarizeY");
          fname = dirname + "polarizationZ" + std::to_string(i);
          VTI::writeDataScalar(polarizationZ, voxel, fname.c_str(), "polarizeZ");
        }
#endif

#ifdef PROFILING
        {
          END_TIMER(TIMERS::POLARIZATION)
          START_TIMER(TIMERS::FFT)
        }
#endif



        /** FFT Computation **/
        result = performFFT(d_polarizationX,plan);
        if (result != HIPFFT_SUCCESS) {
          std::cout << "CUFFT failed with result " << result << "\n";
          #pragma omp cancel parallel
          exit(EXIT_FAILURE);
        }

        result = performFFT(d_polarizationY,plan);
        if (result != HIPFFT_SUCCESS) {
          std::cout << "CUFFT failed with result " << result << "\n";
          #pragma omp cancel parallel
          exit(EXIT_FAILURE);
        }

        result = performFFT(d_polarizationZ,plan);
        if (result != HIPFFT_SUCCESS) {
          std::cout << "CUFFT failed with result " << result << "\n";
          exit(EXIT_FAILURE);
        }

        FFTIgor<<<BlockSize,NUM_THREADS>>>(d_polarizationX, vx);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        FFTIgor<<<BlockSize,NUM_THREADS>>>(d_polarizationY, vx);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

        FFTIgor<<<BlockSize,NUM_THREADS>>>(d_polarizationZ, vx);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

#ifdef DUMP_FILES
        CUDA_CHECK_RETURN(hipMemcpy(polarizationX,
                                     d_polarizationX,
                                     sizeof(Complex) * voxelSize,
                                     hipMemcpyDeviceToHost));
        gpuErrchk(hipPeekAtLastError());
        CUDA_CHECK_RETURN(hipMemcpy(polarizationY,
                                     d_polarizationY,
                                     sizeof(Complex) * voxelSize,
                                     hipMemcpyDeviceToHost));
        gpuErrchk(hipPeekAtLastError());
        CUDA_CHECK_RETURN(hipMemcpy(polarizationZ,
                                     d_polarizationZ,
                                     sizeof(Complex) * voxelSize,
                                     hipMemcpyDeviceToHost));
        gpuErrchk(hipPeekAtLastError());
        {
          std::string dirname = "FFT/";
          std::string fname = dirname + "polarizationXfft" + std::to_string(i);
          VTI::writeDataScalar(polarizationX, voxel, fname.c_str(), "polarizeXfft");
          fname = dirname + "polarizationYfft" + std::to_string(i);
          VTI::writeDataScalar(polarizationY, voxel, fname.c_str(), "polarizeYfft");
          fname = dirname + "polarizationZfft" + std::to_string(i);
          VTI::writeDataScalar(polarizationZ, voxel, fname.c_str(), "polarizeZfft");
        }
#endif

#ifdef PROFILING
        {
          END_TIMER(TIMERS::FFT)
          START_TIMER(TIMERS::SCATTER3D)
        }
#endif



        /** Scatter 3D computation **/
        computeScatter3D <<< BlockSize, NUM_THREADS >>> (d_polarizationX, d_polarizationY, d_polarizationZ, d_scatter3D, eleField, voxelSize, vx, idata.physSize,
                                                         idata.if2DComputation());
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());

#ifdef DUMP_FILES
        CUDA_CHECK_RETURN(hipMemcpy(scatter3D, d_scatter3D, sizeof(Real) * voxelSize, hipMemcpyDeviceToHost));
        gpuErrchk(hipPeekAtLastError())
        {
          std::string dirname = "Scatter/";
          std::string fname = dirname + "scatter" + std::to_string(i);
          VTI::writeDataScalar(scatter3D, voxel, fname.c_str(), "scatter3D");
        }

#endif

#ifdef PROFILING
        {
          END_TIMER(TIMERS::SCATTER3D)
          START_TIMER(TIMERS::EWALDS)
        }
#endif

#ifdef EOC
        CUDA_CHECK_RETURN(hipMemcpy(scatter3D, d_scatter3D, sizeof(Real) * voxelSize, hipMemcpyDeviceToHost));
        gpuErrchk(hipPeekAtLastError());

#ifdef PROFILING
        {

        }
#endif
        computeEwaldProjectionCPU(projectionCPU, scatter3D, vx, eleField.k.x);
#else
        hipMemset(d_rotProjection, 0, voxel[0] * voxel[1] * sizeof(Real));

        computeEwaldProjectionGPU <<< BlockSize2, NUM_THREADS >>> (d_projection,d_rotProjection, d_scatter3D, vx,
            eleField.k.z,idata.physSize, static_cast<Interpolation::EwaldsInterpolation>(idata.ewaldsInterpolation),
                                                                   idata.if2DComputation());
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
#ifdef PROFILING
        {
          END_TIMER(TIMERS::EWALDS)
          START_TIMER(TIMERS::ROTATION)
        }
#endif
        const double alpha = cos(angle);
        const double beta = sin(angle);

        /**https://docs.opencv.org/2.4/modules/imgproc/doc/geometric_transformations.html?highlight=warpaffine**/
        const double coeffs[2][3]{
            alpha, beta, static_cast<Real>(((1 - alpha) * voxel[0] / 2 - beta * voxel[1] / 2.)),
            -beta, alpha, static_cast<Real>(beta * voxel[0] / 2. + (1 - alpha) * voxel[1] / 2.)
        };

#ifdef DOUBLE_PRECISION
        NppStatus status = nppiWarpAffine_64f_C1R(d_projection,
                                                  sizeImage,
                                                  voxel[1] * sizeof(Real),
                                                  rect,
                                                  d_rotProjection,
                                                  voxel[1] * sizeof(Real),
                                                  rect,
                                                  coeffs,
                                                  NPPI_INTER_LINEAR);

#else
        NppStatus status = nppiWarpAffine_32f_C1R(d_projection,
                                                  sizeImage,
                                                  voxel[1] * sizeof(Real),
                                                  rect,
                                                  d_rotProjection,
                                                  voxel[1] * sizeof(Real),
                                                  rect,
                                                  coeffs,
                                                  NPPI_INTER_LINEAR);
#endif

        if (status < 0) {
          std::cout << "Image rotation failed with error = " << status << "\n";
          exit(-1);
        }
        if(status != NPP_SUCCESS){
          std::cout << "[WARNING] Image rotation warning = " << status << "\n";
        }

        if(idata.rotMask){
          computeRotationMask<<< BlockSize2, NUM_THREADS >>>(d_rotProjection,d_mask,vx);
          hipDeviceSynchronize();
        }

        const Real factor = static_cast<Real>(1.0);
#ifdef DOUBLE_PRECISION
        stat = hipblasDaxpy(handle, voxel[0] * voxel[1], &factor, d_rotProjection, 1, d_projectionAverage, 1);
#else
        stat = hipblasSaxpy(handle, voxel[0] * voxel[1], &factor, d_rotProjection, 1, d_projectionAverage, 1);
#endif
        if (stat != HIPBLAS_STATUS_SUCCESS) {
          std::cout << "CUBLAS during sum failed  with status " << stat << "\n";
          exit(EXIT_FAILURE);
        }

#ifdef PROFILING
        {
          END_TIMER(TIMERS::ROTATION)
        }
#endif
#endif
      }

      if(idata.rotMask){
        averageRotation<<<BlockSize2,NUM_THREADS>>>(d_projectionAverage,d_mask,vx);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
      }
      else {
        /// The averaging out for all angles
        const Real alphaFac = static_cast<Real>(1.0 / numAnglesRotation);
#ifdef DOUBLE_PRECISION
        stat = hipblasDscal(handle, voxel[0] * voxel[1], &alphaFac, d_projectionAverage, 1);
#else
        stat = hipblasSscal(handle, voxel[0] * voxel[1], &alphaFac, d_projectionAverage, 1);
#endif
        if (stat != HIPBLAS_STATUS_SUCCESS) {
          std::cout << "CUBLAS during averaging failed  with status " << stat << "\n";
          exit(EXIT_FAILURE);
        }
      }
#ifdef PROFILING
      {
        START_TIMER(TIMERS::MEMCOPY_GPU_CPU)
      }
#endif
      CUDA_CHECK_RETURN(hipMemcpy(&projectionGPUAveraged[j * voxel[0] * voxel[1]],
                                   d_projectionAverage,
                                   sizeof(Real) * (voxel[0] * voxel[1]),
                                   hipMemcpyDeviceToHost));
      gpuErrchk(hipPeekAtLastError());

#ifdef PROFILING
      {
        END_TIMER(TIMERS::MEMCOPY_GPU_CPU)
        END_TIMER(TIMERS::ENERGY)
      }
#endif
    }

    /** Freeing bunch of memories not required now **/
    CUDA_CHECK_RETURN(hipFree(d_polarizationZ));
    gpuErrchk(hipPeekAtLastError());
    CUDA_CHECK_RETURN(hipFree(d_polarizationY));
    gpuErrchk(hipPeekAtLastError());
    CUDA_CHECK_RETURN(hipFree(d_polarizationX));
    gpuErrchk(hipPeekAtLastError());
    CUDA_CHECK_RETURN(hipFree(d_voxelInput));
    gpuErrchk(hipPeekAtLastError());

#ifndef EOC
    CUDA_CHECK_RETURN(hipFree(d_projection));
    gpuErrchk(hipPeekAtLastError());
    CUDA_CHECK_RETURN(hipFree(d_rotProjection));
    gpuErrchk(hipPeekAtLastError());
    if(idata.rotMask) {
      CUDA_CHECK_RETURN(hipFree(d_mask));
      gpuErrchk(hipPeekAtLastError());
    }
#endif

    delete[] polarizationX;
    delete[] polarizationY;
    delete[] polarizationZ;
    delete[] scatter3D;


    hipfftDestroy(plan);
    hipblasDestroy(handle);

  }



#ifdef PROFILING
  std::cout << "\n\n[INFO] Timings Info\n";
  for(int i = 0; i < TIMERS::MAX; i++){
    std::cout << "[TIMERS] " << std::left << std::setw(20) << timersName[i] << ":" << timings[i] << " s\n";
  }
  std::cout << "\n\n";
#endif

#ifdef EOC
  delete[] projectionCPU;
#else
#endif

  return (EXIT_SUCCESS);
}

